#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void dkernel(){

	for(int i=0;i<10;i++){ printf("Hello World\n"); }
}

int main(){
	dkernel<<<1,1>>>();
	hipDeviceSynchronize();
	return 0;
}
