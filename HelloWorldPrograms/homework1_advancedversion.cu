#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 10

__global__ void dkernel(){
	printf("Hello World\n");
}

int main(){
	dkernel<<<1,N>>>();
	hipDeviceSynchronize();
}
