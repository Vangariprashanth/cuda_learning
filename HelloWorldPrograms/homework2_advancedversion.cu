#include<stdio.h>
#include<hip/hip_runtime.h>
# define N 10 

__global__ void dkernel(){
	printf("%u\n",threadIdx.x * threadIdx.x);
}

int main(){
	dkernel<<<1,N>>>();
	hipDeviceSynchronize();
	return 0;
}
