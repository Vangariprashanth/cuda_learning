#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void dkernel(){
	printf("Hello World\n");
}

int main(){
	dkernel<<<1,1>>>();
	printf("CPU 1\n");
	dkernel<<<1,1>>>();
	printf("CPU 2\n");
	dkernel<<<1,1>>>();
	printf("CPU 3\n");
	hipDeviceSynchronize();
	printf("On CPU final call");
	return 0;
}
