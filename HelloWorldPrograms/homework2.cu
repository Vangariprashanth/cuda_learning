//Print the squares of the N numbers

#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 10

__global__ void dkernel(){
	for(int i=1;i<=N;i++){ printf("%d\n",i*i); }
}

int main(){
	dkernel<<<1,1>>>();
	hipDeviceSynchronize();
	return 0;
}
