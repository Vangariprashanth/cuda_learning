#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void dkernel1(){

	printf("Kernel 1: Hello World\n");
}

__global__ void dkernel2(){

	printf("Kernel 2: Hello World\n");
}

int main(){
	dkernel1<<<1,1>>>();
	dkernel2<<<1,1>>>();
	printf("On CPU\n");
	hipDeviceSynchronize();
	return 0;
}
