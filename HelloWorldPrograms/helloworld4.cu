#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void dkernel1(){

	printf("Kernel1: Hello World\n");

}

__global__ void dkernel2(){

	printf("Kernel2: Hello World\n");

}

int main(){
	dkernel1<<<1,1>>>();
	dkernel2<<<1,1>>>();
	hipDeviceSynchronize();
	printf("On CPU\n");
	return 0;
}
